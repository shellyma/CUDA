#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstdio>
#include <math.h>

// Assertion to check for errors
#define CUDA_SAFE_CALL(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
  if (code != hipSuccess){
    fprintf(stderr,"CUDA_SAFE_CALL: %s %s %d\n", hipGetErrorString(code), file, line);
  if (abort) exit(code);
   }
}
#define LOOP    2000
#define TOL     1e-6
#define ARR_LEN 2000
#define NUM_THREAD 16
__global__ void kernel_sor(float *d_A, float *d_B) {
    int index_x = blockIdx.x * blockDim.x + threadIdx.x;
    int index_y = blockIdx.y * blockDim.y + threadIdx.y;

    int grid_width = gridDim.x * gridDim.y* blockDim.x*blockDim.y;
    int index = index_y * grid_width + index_x;

    // check boundaries
    if((index_x > 0) && (index_y > 0) && (index_x < ARR_LEN-1) && (index_y < ARR_LEN-1))
        d_B[index] = d_A[index-1]
		   + d_A[index+1]
		   + d_A[index+ARR_LEN]
		   + d_A[index-ARR_LEN];

}


main (int argc, char **argv) {

    float A[ARR_LEN][ARR_LEN], B[ARR_LEN][ARR_LEN];
    float *d_A, *d_B; // These are the copies of A and B on the GPU
    float *h_B; // This is a host copy of the output of B from the GPU to CPU
    int i, j,iter;
    int num_bytes = ARR_LEN * ARR_LEN * sizeof(float);
    int errCount = 0;
    // Input is randomly generated
	for(i=0;i<ARR_LEN;i++) {
             for(j=0;j<ARR_LEN;j++) {
           	 A[i][j] = (float) rand()/1234;
            
       	      }
         }
   

    hipEvent_t start_event0, stop_event0;
    float elapsed_time0;
    CUDA_SAFE_CALL( hipEventCreate(&start_event0) );
    CUDA_SAFE_CALL( hipEventCreate(&stop_event0) );
    hipEventRecord(start_event0, 0);
    // CPU compuation
    for(iter=1;iter<LOOP;iter++){
      for(i=1;i<ARR_LEN-1;i++) {
        for(j=1;j<ARR_LEN-1;j++) {
            B[i][j] = A[i-1][j]+A[i+1][j]+A[i][j-1]+A[i][j+1];
        }
      }
    }
    hipEventRecord(stop_event0, 0);
    hipEventSynchronize(stop_event0);
    CUDA_SAFE_CALL( hipEventElapsedTime(&elapsed_time0,start_event0, stop_event0) );



    h_B = (float *)malloc(num_bytes);
    memset(h_B, 0, num_bytes);
    //ALLOCATE MEMORY FOR GPU COPIES OF A AND B
    hipMalloc((void**)&d_A, num_bytes);
    hipMalloc((void**)&d_B, num_bytes);
    hipMemset(d_A, 0, num_bytes);
    hipMemset(d_B, 0, num_bytes);

    //COPY A TO GPU
    hipMemcpy(d_A, A, num_bytes, hipMemcpyHostToDevice);

    hipEvent_t start_event, stop_event;
    float elapsed_time;
    CUDA_SAFE_CALL( hipEventCreate(&start_event) );
    CUDA_SAFE_CALL( hipEventCreate(&stop_event) );
    hipEventRecord(start_event, 0);
    
    dim3 block_size(NUM_THREAD,NUM_THREAD);
   // dim3 grid_size(ARR_LEN/block_size.x,ARR_LEN/block_size.y,1);
   dim3 grid_size(2,8);    
//Multiple iterations controlled by CPU
    for(iter = 0; iter <LOOP; iter++){
       kernel_sor<<<grid_size,block_size>>>(d_A,d_B);
       hipDeviceSynchronize();
    }

    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    CUDA_SAFE_CALL( hipEventElapsedTime(&elapsed_time,start_event, stop_event) );

    //COPY B BACK FROM GPU
    hipMemcpy(h_B, d_B, num_bytes, hipMemcpyDeviceToHost);

    //TODO: Compare results
    int index = iter + i*ARR_LEN;
    for(iter = 0; iter < ARR_LEN; iter++){
       for(i=0; i< ARR_LEN; i ++){
	 if(h_B[index] - B[iter][i] > TOL){
          errCount ++; 
         }
      }
    }
    //Time compare
    printf("Error Count: \t%d\n",errCount);
    printf("CPU computation time: \t%.2f ms\n",elapsed_time0);
    printf("GPU calculation time:  \t%.2f ms\n", elapsed_time);
    printf("CUDA Speedup:\t%.2fx\n",(elapsed_time0/elapsed_time));

    hipFree(d_A);
    hipFree(d_B);
    free(h_B);
}
